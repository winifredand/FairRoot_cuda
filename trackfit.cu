////均采用CUDA7.5的函数编写；////////////
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <math.h>
#include "HitTrk.h"
#include "trackfit_kernel.cu"

///////////////内核函数////////////
extern "C" void CircleFitG(double X[HIT], double Y[HIT], double Z[HIT], double Zerr[HIT], double *Mx, double *My, double *M0, double *result)
{
	////////////在设备段分配显存,d_表示device,////////
	double *d_X;
	double *d_Y;
	double *d_Z;
	double *d_Zerr;
	double *d_Mx;
	double *d_My;
	double *d_MO;
	double *d_result;

	hipError_t cudaStatus; //状态监测；

	///////////////分配设备/////////////////
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//////////分配内存/////////
	size_t size = sizeof(double);
	checkCudaErrors(hipMalloc((void **)&d_X, size*HIT));
	checkCudaErrors(hipMalloc((void **)&d_Y, size*HIT));
	checkCudaErrors(hipMalloc((void **)&d_Z, size*HIT));
	checkCudaErrors(hipMalloc((void **)&d_Zerr, size*HIT));
	checkCudaErrors(hipMalloc((void **)&d_My, size));
	checkCudaErrors(hipMalloc((void **)&d_MO, size));
	checkCudaErrors(hipMalloc((void **)&d_Mx, size));
	checkCudaErrors(hipMalloc((void **)&d_result, size * 8));

	//////将内存中的数据读入显存，完成主机即CPU对CUDA设备的数据写入/////
	checkCudaErrors(hipMemcpy(d_X, X, size*HIT, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Y, Y, size*HIT, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Z, Z, size*HIT, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Zerr, Zerr, size*HIT, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Mx, Mx, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_My, My, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_MO, M0, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_result, result, size * 8, hipMemcpyHostToDevice));

	////////设置运行参数，即网格的形状和线程块的形状///////
	//    unsigned int threads = HIT;
	//    unsigned int tracks = TRK;
	dim3 dimBlock2(HIT, 1);// HIT=20,此句表示每个block有20个线程；
	dim3 dimGrid2(1, 1); //表示1*1个block；

	 /////////时间函数///////////
	hipEvent_t start, stop;
	float time_kernel;
	checkCudaErrors(hipEventRecord(start, 0));//开始计时；

	///////调用内核函数进行计算////////////////
	Fit <<< dimGrid2, dimBlock2 >>> (d_X, d_Y, d_Z, d_Zerr, d_Mx, d_My, d_MO, d_result);

	checkCudaErrors(hipEventRecord(stop, 0));//结束计时；
	checkCudaErrors(hipEventSynchronize(start));
	checkCudaErrors(hipEventSynchronize(stop));

	checkCudaErrors(hipEventElapsedTime(&time_kernel, start, stop));//计算时间差；

	checkCudaErrors(hipEventDestroy(start));//destory the event
	checkCudaErrors(hipEventDestroy(stop));

	printf("kernel:\t\t%.2f\n", time_kernel);//输出内核函数运行时间；

	//////////////将结果从显存设备段写入内存主机端/////////////
	checkCudaErrors(hipMemcpy(result, d_result, size * 8, hipMemcpyDeviceToHost));

	/////////////打印结果/////////
	printf("%f, %f, %f, %f, %f, %f, %f, %f,\n",result[0],result[1],result[2],result[3],result[4],result[5],result[6],result[7]);

	/////////////释放显存////////
Error:
	checkCudaErrors(hipFree(d_X));
	checkCudaErrors(hipFree(d_Y));
	checkCudaErrors(hipFree(d_Z));
	checkCudaErrors(hipFree(d_Zerr));
	checkCudaErrors(hipFree(d_Mx));
	checkCudaErrors(hipFree(d_My));
	checkCudaErrors(hipFree(d_MO));
	checkCudaErrors(hipFree(d_result));
}



extern "C" void CircleFitGAllD(double X[TRK*HIT], double Y[TRK*HIT], double Z[TRK*HIT], double Zerr[TRK*HIT], double Mx[TRK], double My[TRK], double M0[TRK], double result[TRK * 8])
{
	//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );
	/* for(int j=0; j<100; j++){
	for(int i=0; i<HIT; i++){
	printf("%d  Zerr[%i] =  %g  :  Z[%i] =  %g  \n", j ,i, Zerr[i+HIT*j],i ,Z[i+HIT*j]  );
	}
	}
	*/
	double *d_X;
	double *d_Y;
	double *d_Z;
	double *d_Zerr;
	double *d_Mx;
	double *d_My;
	double *d_M0;
	double *d_result;

	/*   result[0]=1;
	result[1]=1;
	result[2]=1;*/

	size_t size = sizeof(double);
	//allocate memory for arrays on device 
	CUDA_SAFE_CALL(hipMalloc((void **)&d_X, size*HIT*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Y, size*HIT*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Z, size*HIT*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Zerr, size*HIT*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Mx, size*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_My, size*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_M0, size*TRK));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_result, size * 8 * TRK));



	CUDA_SAFE_CALL(hipMemcpy(d_X, X, size*HIT*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Y, Y, size*HIT*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Z, Z, size*HIT*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Zerr, Zerr, size*HIT*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Mx, Mx, size*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_My, My, size*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_M0, M0, size*TRK, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_result, result, size * 8 * TRK, hipMemcpyHostToDevice));


	int threads = HIT;
	int tracks = TRK;
	dim3 dimBlock2(threads, 1);
	dim3 dimGrid2(tracks, 1);

	// FitAllD<<< dimGrid2, dimBlock2 >>> (d_X, d_Y,d_Z, d_Zerr,d_Mx,d_My,d_M0, d_result);

	// hipDeviceSynchronize();
	printf(" Now calling the device  \n");
	CUDA_SAFE_CALL(hipMemcpy(result, d_result, size * 8 * TRK, hipMemcpyDeviceToHost));

	/*  for(int j=0; j<tracks; j++){
	printf("%d   ",j);
	for(int i=0; i<8; i++){
	printf("  %f ", result[i+8*j]);
	}
	printf(" \n");
	}
	*/
	//   printf(" Now cleaning device memory  \n");
	CUDA_SAFE_CALL(hipFree(d_X));
	CUDA_SAFE_CALL(hipFree(d_Y));
	CUDA_SAFE_CALL(hipFree(d_Z));
	CUDA_SAFE_CALL(hipFree(d_Zerr));
	CUDA_SAFE_CALL(hipFree(d_Mx));
	CUDA_SAFE_CALL(hipFree(d_My));
	CUDA_SAFE_CALL(hipFree(d_M0));
	CUDA_SAFE_CALL(hipFree(d_result));
	//   printf(" Finish cleaning device memory  \n");
}



extern "C" void CircleFitGAllF(float X[TRK*HIT], float Y[TRK*HIT], float Z[TRK*HIT], float Zerr[TRK*HIT], float Mx[TRK], float My[TRK], float M0[TRK], float result[8 * TRK])
{
	//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );
	/* for(int j=0; j<100; j++){
	for(int i=0; i<HIT; i++){
	printf("%d  Zerr[%i] =  %g  :  Z[%i] =  %g  \n", j ,i, Zerr[i+HIT*j],i ,Z[i+HIT*j]  );
	}
	}
	*/
	float *d_X;
	float *d_Y;
	float *d_Z;
	float *d_Zerr;
	float *d_Mx;
	float *d_My;
	float *d_M0;
	float *d_result;

	/*   result[0]=1;
	result[1]=1;
	result[2]=1;*/

	size_t size = sizeof(float);
	//allocate memory for arrays on device 
	hipMalloc((void **)&d_X, size*HIT*TRK);
	hipMalloc((void **)&d_Y, size*HIT*TRK);
	hipMalloc((void **)&d_Z, size*HIT*TRK);
	hipMalloc((void **)&d_Zerr, size*HIT*TRK);
	hipMalloc((void **)&d_Mx, size*TRK);
	hipMalloc((void **)&d_My, size*TRK);
	hipMalloc((void **)&d_M0, size*TRK);
	hipMalloc((void **)&d_result, size * 8 * TRK);



	hipMemcpy(d_X, X, size*HIT*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, size*HIT*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_Z, Z, size*HIT*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_Zerr, Zerr, size*HIT*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_Mx, Mx, size*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_My, My, size*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_M0, M0, size*TRK, hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, size * 8 * TRK, hipMemcpyHostToDevice);


	int threads = HIT;
	int tracks = TRK;
	dim3 dimBlock2(threads, 1);
	dim3 dimGrid2(tracks, 1);

	// FitAllF<<< dimGrid2, dimBlock2 >>> (d_X, d_Y,d_Z, d_Zerr,d_Mx,d_My,d_M0, d_result);

	//   hipDeviceSynchronize();

	hipMemcpy(result, d_result, size * 8 * TRK, hipMemcpyDeviceToHost);

	/*  for(int j=0; j<tracks; j++){
	printf("%d   ",j);
	for(int i=0; i<8; i++){
	printf("  %f ", result[i+8*j]);
	}
	printf(" \n");
	}
	*/
	// printf(" Now cleaning device memory  \n");
	CUDA_SAFE_CALL(hipFree(d_X));
	CUDA_SAFE_CALL(hipFree(d_Y));
	CUDA_SAFE_CALL(hipFree(d_Z));
	CUDA_SAFE_CALL(hipFree(d_Zerr));
	CUDA_SAFE_CALL(hipFree(d_Mx));
	CUDA_SAFE_CALL(hipFree(d_My));
	CUDA_SAFE_CALL(hipFree(d_M0));
	CUDA_SAFE_CALL(hipFree(d_result));
	//   printf(" Finish cleaning device memory  \n");
}




extern "C" void CircleFitGF(float X[HIT], float Y[HIT], float Z[HIT], float Zerr[HIT], float *Mx, float *My, float *M0, float *result)
{
	//   printf(" Now in Cuda :  Mx =  %g  :  My =  %g  \n", Mx[0],My[0] );

	/*  for(int i=0; i<50; i++){
	printf(" Zerr[%i] =  %g  :  Z[%i] =  %g  \n", i, Zerr[i],i ,Z[i]  );
	}
	*/
	float *d_X;
	float *d_Y;
	float *d_Z;
	float *d_Zerr;
	float *d_Mx;
	float *d_My;
	float *d_M0;
	float *d_result;

	/*   result[0]=1;
	result[1]=1;
	result[2]=1;*/

	size_t size = sizeof(float);
	//allocate memory for arrays on device 
	CUDA_SAFE_CALL(hipMalloc((void **)&d_X, size*HIT));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Y, size*HIT));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Z, size*HIT));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Zerr, size*HIT));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Mx, size));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_My, size));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_M0, size));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_result, size * 8));



	CUDA_SAFE_CALL(hipMemcpy(d_X, X, size*HIT, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Y, Y, size*HIT, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Z, Z, size*HIT, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Zerr, Zerr, size*HIT, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Mx, Mx, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_My, My, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_M0, M0, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_result, result, size * 8, hipMemcpyHostToDevice));


	dim3 dimBlock2(HIT, 1);
	dim3 dimGrid2(1, 1);

	FitF << < dimGrid2, dimBlock2 >> > (d_X, d_Y, d_Z, d_Zerr, d_Mx, d_My, d_M0, d_result);


	CUDA_SAFE_CALL(hipMemcpy(result, d_result, size * 8, hipMemcpyDeviceToHost));

	//  printf(" %f      %f      %f  %   f      %f     %f     %f       %f \n", result[0], result[1],result[2], result[3] ,result[4],result[5] ,result[6],result[7]);

	CUDA_SAFE_CALL(hipFree(d_X));
	CUDA_SAFE_CALL(hipFree(d_Y));
	CUDA_SAFE_CALL(hipFree(d_Z));
	CUDA_SAFE_CALL(hipFree(d_Zerr));
	CUDA_SAFE_CALL(hipFree(d_Mx));
	CUDA_SAFE_CALL(hipFree(d_My));
	CUDA_SAFE_CALL(hipFree(d_M0));
	CUDA_SAFE_CALL(hipFree(d_result));

}
